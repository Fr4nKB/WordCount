#include <hip/hip_runtime.h>


#include <hiprand/hiprand_kernel.h>

#include <time.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <unordered_map>
#include <algorithm>
#include <iterator>

using namespace std;

int NBLOCK = 1, NTHREAD = 1, MAX_ELEM = 1;

__global__ void initRNG(hiprandState* const rngStates, const unsigned int seed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &rngStates[tid]);
}


__global__ void init_array(int *words_int, int *totlen, int *var, hiprandState* const rngStates) {
    int end = *totlen;
    int range = *var;

    for(int i = 0; i < end; i++) {
        words_int[i] = int(hiprand(&rngStates[0])%range);
    }

}

__global__ void thread_func(int *words_int, int *nwords, int *totlen, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int wordsPerThread = *nwords;
    int start = tid*wordsPerThread;
    int end = (tid+1)*wordsPerThread;
    int size = *totlen;

    for(int i = start; i < end; i++) {
        if(i < size) atomicAdd(&result[words_int[i]], 1);
    }

}

//launches 'Nthreads' and gives to each one of them a portion of 'words'
vector<double> mainthread(int totlen, int *array_len, int *d_words, int *d_wordsXthread, int *d_results) {

    int *h_results = new int[totlen];
    hipMemset(d_results, 0, sizeof(int) * totlen);
    
    clock_t startTime = clock();

    thread_func<<<NBLOCK, NTHREAD>>>(d_words, d_wordsXthread, array_len, d_results);
	hipDeviceSynchronize();

    clock_t countingTime = clock();
    
    hipMemcpy(h_results, d_results, sizeof(int) * totlen, hipMemcpyDeviceToHost);

    clock_t endTime = clock();

    delete[] h_results;

    vector<double> times;
    times.push_back((countingTime - startTime)/1000.0);
    times.push_back((endTime - startTime)/1000.0);

    return times;
}

void benchmark(int totlen, int variabilty, int warps, int n_iter) {
    int h_wordsXthread, *d_wordsXthread, *d_words, *d_results, *var, *array_len;
    vector<vector<double>>avg_res;
    vector<double>avg(2);
	hiprandState* devStates;

    MAX_ELEM = variabilty;

    hipHostMalloc((void**)&d_words, sizeof(int) * totlen, hipHostMallocDefault);
    hipMalloc((void**)&d_wordsXthread, sizeof(int));
    hipMalloc((void**)&d_results, sizeof(int) * totlen);
    hipMalloc((void**)&var, sizeof(int));
    hipMalloc((void**)&array_len, sizeof(int));
	hipMalloc((void**)&devStates, sizeof(hiprandState));

    hipMemcpy(var, &MAX_ELEM, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(array_len, &totlen, sizeof(int), hipMemcpyHostToDevice);
    
    initRNG<<<1, 1>>>(devStates, time(NULL));
    init_array<<<1, 1>>>(d_words, array_len, var, devStates);

    for(int i = 0; NTHREAD < warps*32; i++) {

        NTHREAD = 32*pow(2,i);
        h_wordsXthread = ceil(((float) totlen)/(float)(NBLOCK * NTHREAD));
        hipMemcpy(d_wordsXthread, &h_wordsXthread, sizeof(int), hipMemcpyHostToDevice);
        avg[0] = avg[1] = 0;
        
        cout<<"STARTING THREADS ("<<NBLOCK<<", "<<NTHREAD<<")"<<endl;
        for(int j = 0; j < n_iter; j++) {
            cout<<"Run: "<<j+1<<endl;
            auto tmp = mainthread(totlen, array_len, d_words, d_wordsXthread, d_results);
            avg[0] += tmp[0]/n_iter; avg[1] += tmp[1]/n_iter;
        }
        avg_res.push_back(avg);

    }

    hipFree(d_words);
    hipFree(d_wordsXthread);
    hipFree(d_results);
    hipFree(var);
    hipFree(devStates);

    fstream output;
    output.open("output_v3_b"+to_string(NBLOCK)+"_n_"+to_string(totlen)+".txt", ios::out);
    for(int i = 0; i < avg_res.size(); i++) {
        output<<avg_res[i][0]<<"\t\t"<<avg_res[i][1]<<endl;
    }
    output.close();

}

// arrayDim maxElem nblocks nthreads n_iter
int main(int argc, char* argv[]) {

    if(argc != 6) return -1;
    NBLOCK = stoi(argv[3]);
    benchmark(stoi(argv[1]), stoi(argv[2]), stoi(argv[4]), stoi(argv[5]));

    return 0;
}